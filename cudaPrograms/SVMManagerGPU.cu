//
// Created by cristobal, 2018
//

#include <cstddef>
#include "SVMManagerGPU.cuh"
#include "defs.h"


__host__ SVMManagerGPU::SVMManagerGPU(float* SVMWeights, size_t size) : weightsArraySize(size) {
  size_t totalBytes = size * sizeof(float);


  cf = hipCreateChannelDesc<float>();

  gpuErrchk(hipMallocArray(&dSVMWeights, &cf, size, 1));
  gpuErrchk(hipMemcpyToArray(dSVMWeights, 0, 0, SVMWeights, totalBytes, hipMemcpyHostToDevice));
  gpuErrchk(hipBindTextureToArray(texRef, dSVMWeights, cf));


}

__host__ SVMManagerGPU::~SVMManagerGPU() {
  gpuErrchk(hipUnbindTexture(texRef));
  gpuErrchk(hipFree(dSVMWeights));
}

size_t SVMManagerGPU::getWeightsArraySize() {
  return weightsArraySize;
}

void* SVMManagerGPU::getDeviceArray() {
  return (void*) dSVMWeights;
}
